#include "hip/hip_runtime.h"
#include "constants.h"

#include <hiprand.h>
#include <iostream>
#include <ctime>

#include "hip/hip_runtime_api.h"

#include "dataGeneration.h"
#include "dataModifying.cuh"

spheres generateSpheres()
{
	spheres data;

	// alokuje pami�� na GPU
	checkCudaErrors(hipMalloc(&(data.centers), sphere_count * sizeof(float3)));
	checkCudaErrors(hipMalloc(&(data.radius), sphere_count * sizeof(float)));
	checkCudaErrors(hipMalloc(&(data.colors), sphere_count * sizeof(float3)));

	// tworzy generator cuRand
	hiprandGenerator_t gen;
	curandCheck(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
	curandCheck(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));

	// losuje warto�ci przy u�yciu biblioteki cuRand
	curandCheck(hiprandGenerateUniform(gen, (float*)data.centers, 3 * sphere_count));
	curandCheck(hiprandGenerateUniform(gen, data.radius, sphere_count));
	curandCheck(hiprandGenerateUniform(gen, (float*)data.colors, 3 * sphere_count));

	// niszczy generator po zako�czeniu losowania
	curandCheck(hiprandDestroyGenerator(gen));

	// modyfikuje wylosowane dane
	modifyData(&data);

	// zwraca gotowe dane
	return data;
}

lights generateLights(int count)
{
	if (count == 0) return lights{};

	lights lights;
	lights.count = count;

	constexpr float radius = 100.0f;

	// alokuje pami�� GPU na �wiat�a
	checkCudaErrors(hipMalloc(&(lights.x), count * sizeof(float)));
	checkCudaErrors(hipMalloc(&(lights.y), count * sizeof(float)));
	checkCudaErrors(hipMalloc(&(lights.z), count * sizeof(float)));

	checkCudaErrors(hipMalloc(&(lights.r), count * sizeof(float)));
	checkCudaErrors(hipMalloc(&(lights.g), count * sizeof(float)));
	checkCudaErrors(hipMalloc(&(lights.b), count * sizeof(float)));

	// tworzy generator cuRand
	hiprandGenerator_t gen;
	curandCheck(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
	curandCheck(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));

	// alokuje pami�� hosta na pozycje �wiate�
	float* x = new float[count];
	float* y = new float[count];
	float* z = new float[count];

	// sekwencyjnie wype�nia kolejne �wiat�a, rozmieszczaj�c je r�wnomiernie na okr�gu o promieniu 100
	x[0] = 0.0f; y[0] = 0.0f; z[0] = radius;

	const float sin_d_phi = sinf(6.28f / count);
	const float cos_d_phi = cosf(6.28f / count);

	for (int i = 1; i < count; i++)
	{
		x[i] = x[i - 1] * cos_d_phi - z[i - 1] * sin_d_phi;
		y[i] = 0.0f;
		z[i] = x[i - 1] * sin_d_phi + z[i - 1] * cos_d_phi;
	}

	// kopiuje wype�nione dane do pami�ci GPU
	checkCudaErrors(hipMemcpy(lights.x, x, count * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(lights.y, y, count * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(lights.z, z, count * sizeof(float), hipMemcpyHostToDevice));

	// usuwa bufory ze strony hosta
	delete x;
	delete y;
	delete z;

	// losuje kolory �wiate�
	curandCheck(hiprandGenerateUniform(gen, lights.r, count));
	curandCheck(hiprandGenerateUniform(gen, lights.g, count));
	curandCheck(hiprandGenerateUniform(gen, lights.b, count));

	// niszczy generator cuRand
	curandCheck(hiprandDestroyGenerator(gen));

	return lights;
}